#include "hip/hip_runtime.h"
/*=============================================================================

  PYVINECOPULIB: A python interface to vinecopulib.

  Copyright (c) University College London (UCL). All rights reserved.

  This software is distributed WITHOUT ANY WARRANTY; without even
  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
  PURPOSE.

  See LICENSE.txt in the top level directory for details.

=============================================================================*/

#include "catch.hpp"
#include "pvCatchMain.h"
#include <iostream>
#include <thrust/version.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include <time.h>

TEST_CASE( "Check Thrust Exists", "[CUDA]" ) {
  int major = THRUST_MAJOR_VERSION;
  int minor = THRUST_MINOR_VERSION;
  std::cout << "Thrust v" << major << "." << minor << std::endl;
  REQUIRE(major > 0);
  REQUIRE(minor > 0);
}

TEST_CASE( "Copy to device, sort", "[CUDA]" ) {
  thrust::host_vector<int> h(2);
  h[0] = 3;
  h[1] = 1;
  thrust::device_vector<int> d = h;
  thrust::sort(d.begin(), d.end());
  REQUIRE(d[0] == 1);
  REQUIRE(d[1] == 3);
}

TEST_CASE( "Sort 32M numbers on GPU", "[CUDA]" ) {

  // Create test array.
  thrust::host_vector<int> h(32 << 20);
  std::generate(h.begin(), h.end(), rand);

  // Start clock.
  std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

  // Copy to card, compute, copy back.
  thrust::device_vector<int> d = h;
  thrust::sort(d.begin(), d.end());
  thrust::copy(d.begin(), d.end(), h.begin());

  // Stop clock.
  std::chrono::steady_clock::time_point end= std::chrono::steady_clock::now();
  std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() <<std::endl;

  REQUIRE(h.size() == d.size());
}
